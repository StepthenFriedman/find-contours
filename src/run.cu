#include "hip/hip_runtime.h"
#include ""
#include "dbg.cuh"

int main(){
    dbg::find_variance();
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}