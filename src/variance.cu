#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "nvcuvid.h"
//#include <opencv2/core.hpp>
//#include <opencv2/highgui.hpp>

hipError_t getVariance(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei);

__global__ void VarKernal(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y, i, j;
    if (x<(ipt_wid-knl_size+1)&&y<(ipt_hei-knl_size+1)){
        double sum=0.,temp=0.,avg;
        for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) sum+=(double)ipt[(y+j)*ipt_wid+(x+i)];
        avg=sum/(double)(knl_size*knl_size);sum=0.;
        for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) {
            temp=((double)ipt[(y+j)*ipt_wid+(x+i)]-avg);
            sum+=temp*temp;
        }
        opt[y*(ipt_wid-knl_size+1)+x]=sqrt(sum);
    }
}

int main()
{
    const unsigned int ipt_wid=5,ipt_hei=5,knl_size=3;
    unsigned int i,j;
    const int ipt[ipt_hei][ipt_wid]={
                                    1, 2, 3, 4, 5, 
                                    6, 7, 8, 9, 10, 
                                    11,12,13,14,15, 
                                    16,17,18,19,20,
                                    21,22,23,24,25};
    int opt[ipt_hei-knl_size+1][ipt_wid-knl_size+1]={0};

    hipError_t cudaStatus = getVariance( &opt[0][0], &ipt[0][0], knl_size, ipt_wid, ipt_hei);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    for (i=0;i<ipt_hei-knl_size+1;i++) {for (j=0;j<ipt_wid-knl_size+1;j++) printf("%d ",opt[i][j]);putchar('\n');}

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t getVariance(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    int *dev_ipt=NULL,*dev_opt=NULL;
    const unsigned int opt_wid=ipt_wid-knl_size+1,opt_hei=ipt_hei-knl_size+1;
    hipError_t cudaStatus;
    dim3 blocks((opt_wid+31)/32,(opt_hei+31)/32);
    dim3 threadsPerBlock(32,32);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
 
    cudaStatus = hipMalloc((void**)&dev_opt, opt_wid*opt_hei * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_ipt, ipt_wid*ipt_hei * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_ipt, ipt, ipt_wid*ipt_hei * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    VarKernal<<< blocks, threadsPerBlock >>>(dev_opt, dev_ipt, knl_size, ipt_wid, ipt_hei);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "VarKernal launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(opt, dev_opt, opt_wid*opt_hei * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_opt);
    hipFree(dev_ipt);
    
    return cudaStatus;
}