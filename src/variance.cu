#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "lodepng.h"
#include "lodepng.cpp"
//#include "nvcuvid.h"
//#include <opencv2/core.hpp>
//#include <opencv2/highgui.hpp>

hipError_t getVariance(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei);

__global__ void varKernal(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y, i, j;
    if (x<(ipt_wid-knl_size+1)&&y<(ipt_hei-knl_size+1)){
        double sum=0.,temp=0.,avg;
        for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) sum+=(double)ipt[(y+j)*ipt_wid+(x+i)];
        avg=sum/(double)(knl_size*knl_size);sum=0.;
        for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) {
            temp=((double)ipt[(y+j)*ipt_wid+(x+i)]-avg);
            sum+=temp*temp;
        }
        opt[y*(ipt_wid-knl_size+1)+x]=sqrt(sum);
    }
}

__global__ void varKernal3D(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y, i,j,k;
    const unsigned int opt_wid=ipt_wid-knl_size+1,opt_hei=ipt_hei-knl_size+1;
    double sum=0.,temp=0.,avg=0.;
    if (x<(opt_wid)&&y<(opt_hei)) {
        for (k=0;k<3;k++) {
            avg=0.;
            for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) avg+=ipt[((y+i)*ipt_wid+(x+j))*3+k];
            avg/=(double)(knl_size*knl_size);
            for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) temp=avg-ipt[((y+i)*ipt_wid+(x+j))*3+k],sum+=temp*temp;
            opt[(y*opt_wid+x)*3+k]=sum;
        }
        sum/=(double)(knl_size*knl_size)*3;
        opt[(y*opt_wid+x)*3]=opt[(y*opt_wid+x)*3+1]=opt[(y*opt_wid+x)*3+2]=sum;
    }
}

__global__ void poolingKernal(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y, i,j,k;
    const unsigned int opt_wid=ipt_wid-knl_size+1,opt_hei=ipt_hei-knl_size+1;
    double sum=0.,temp=0.,avg=0.;
    if (x<(opt_wid)&&y<(opt_hei)) for (k=0;k<3;k++) {
        avg=0.;
        for (i=0;i<knl_size;i++) for (j=0;j<knl_size;j++) avg+=ipt[((y+i)*ipt_wid+(x+j))*3+k];
        avg/=(double)(knl_size*knl_size);
        opt[(y*opt_wid+x)*3+k]=avg;
    }
}

int dbg2(){
    int i,j,k;
    std::vector<unsigned char> in_image;
    unsigned int ipt_wid,ipt_hei,knl_size=3;
    char error=lodepng::decode(in_image, ipt_wid, ipt_hei, "../resource/Lenna.png",LCT_RGBA);
    const unsigned int opt_wid=ipt_wid-knl_size+1,opt_hei=ipt_hei-knl_size+1;


    printf("size:%lu wid:%u hei:%u\n",in_image.size(),ipt_wid,ipt_hei);
    int* input_image = new int[ipt_wid*ipt_hei*3];
    int* output_image = new int[ipt_wid*ipt_hei*3];
    for (i=0;i<ipt_hei;i++) for (j=0;j<ipt_wid;j++) for (k=0;k<3;k++){
        input_image[(i*ipt_wid+j)*3+k]=in_image[(i*ipt_wid+j)*4+k];
    }
    printf("%s\n",lodepng_error_text(error));

    hipError_t cudaStatus = getVariance(output_image,input_image,knl_size,ipt_wid,ipt_hei);
    std::vector<unsigned char> out_image;
    
    for (i=0;i<opt_hei;i++) for (j=0;j<opt_wid;j++) {
        for (k=0;k<3;k++){
            out_image.push_back(output_image[(i*opt_wid+j)*3+k]);
        }
        out_image.push_back(255);
    }
    printf("ok!\n");
    error= lodepng::encode("../resource/result.png", out_image, opt_wid, opt_hei);
    printf("%s\n",lodepng_error_text(error));
    return 0;
}
//rgb
int dbg3(){
    int i,j,k;
    const unsigned int opt_wid=512,opt_hei=512;

    std::vector<unsigned char> out_image;
    for (i=0;i<opt_hei;i++) for (j=0;j<opt_wid;j++) {
        if (i<100&&j<100){
            out_image.push_back(100);
            out_image.push_back(0);
            out_image.push_back(0);
            out_image.push_back(255);
        }else{
            out_image.push_back(0);
            out_image.push_back(0);
            out_image.push_back(100);
            out_image.push_back(255);
        }
    }
    printf("ok!\n");
    char error= lodepng::encode("../resource/result.png", out_image, opt_wid, opt_hei);
    printf("%s\n",lodepng_error_text(error));
    return 0;
}

int dbg1(){
    const unsigned int ipt_wid=5,ipt_hei=5,knl_size=3;
    unsigned int i,j;
    const int ipt[ipt_hei][ipt_wid]={
                                    1, 2, 3, 4, 5, 
                                    6, 7, 8, 9, 10, 
                                    11,12,13,14,15, 
                                    16,17,18,19,20,
                                    21,22,23,24,25};
    int opt[ipt_hei-knl_size+1][ipt_wid-knl_size+1]={0};

    hipError_t cudaStatus = getVariance( &opt[0][0], &ipt[0][0], knl_size, ipt_wid, ipt_hei);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    for (i=0;i<ipt_hei-knl_size+1;i++) {for (j=0;j<ipt_wid-knl_size+1;j++) printf("%d ",opt[i][j]);putchar('\n');}

    return 0;
}

int main(){
    dbg2();
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

hipError_t getVariance(int *opt, const int *ipt, const unsigned int knl_size, const unsigned int ipt_wid, const unsigned int ipt_hei)
{
    int *dev_ipt=NULL,*dev_opt=NULL;
    const unsigned int opt_wid=ipt_wid-knl_size+1,opt_hei=ipt_hei-knl_size+1;
    hipError_t cudaStatus;
    dim3 blocks((opt_wid+31)/32,(opt_hei+31)/32);
    dim3 threadsPerBlock(32,32);
    printf("ipt:(%d,%d)->%d opt:(%d,%d)->%d\n",ipt_wid,ipt_hei,ipt_wid*ipt_hei*3,opt_wid,opt_hei,opt_wid*opt_hei*3);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
 
    cudaStatus = hipMalloc((void**)&dev_opt, opt_wid*opt_hei*3 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_ipt, ipt_wid*ipt_hei*3 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_ipt, ipt, ipt_wid*ipt_hei*3 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    varKernal3D<<< blocks, threadsPerBlock >>>(dev_opt, dev_ipt, knl_size, ipt_wid, ipt_hei);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "varKernal launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(opt, dev_opt, opt_wid*opt_hei*3 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_opt);
    hipFree(dev_ipt);
    
    return cudaStatus;
}